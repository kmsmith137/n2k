#include "hip/hip_runtime.h"
#include <complex>
#include <iostream>
#include <gputils.hpp>

#include "../include/n2k/Correlator.hpp"
#include "../include/n2k/CorrelatorKernel.hpp"

using namespace std;
using namespace gputils;
using namespace n2k;


// -------------------------------------------------------------------------------------------------
//
// unpack_4bit(), pack_4bit()


// Slow, intended only for testing!
// Given an array of shape S, unpack int32 -> int4[8] and return an array of shape S+(8,)
__host__ Array<int> unpack_4bit(const Array<int> &src)
{
    assert(src.ndim >= 1);
    
    vector<ssize_t> dst_shape(src.ndim+1, 8);
    for (int d = 0; d < src.ndim; d++)
	dst_shape[d] = src.shape[d];

    Array<int> dst(dst_shape, af_rhost);

    for (auto ix = dst.ix_start(); dst.ix_valid(ix); dst.ix_next(ix)) {
	// Extract 4 bits from x, starting at bit b.
	int x = src.at(src.ndim, &ix[0]);
	int b = ix[src.ndim] * 4;
	dst.at(ix) = (x << (28-b)) >> 28;
    }

    return dst;
}


// Slow, intended only for testing!
// Given an array of shape S+(8,), pack int4[8] -> int32 and return an array of shape S.
__host__ Array<int> pack_4bit(const Array<int> &src)
{
    assert(src.ndim >= 2);
    assert(src.shape[src.ndim-1] == 8);

    Array<int> dst(src.ndim-1, &src.shape[0], af_rhost | af_zero);
    
    for (auto ix = src.ix_start(); src.ix_valid(ix); src.ix_next(ix)) {
	// Extract 4 bits from x, starting at bit b.
	int x = src.at(ix);
	int b = ix[src.ndim-1] * 4;
	
	assert ((x >= -8) && (x <= 7));
	dst.at(dst.ndim, &ix[0]) |= ((x & 0xf) << b);
    }

    return dst;
}


__host__ void test_pack_unpack()
{
    int n = 100;
    Array<int> a({n}, af_rhost | af_random);
    Array<int> b = unpack_4bit(a);
    Array<int> c = pack_4bit(b);
    
    assert(b.shape_equals({n,8}));
    assert(c.shape_equals({n}));

#if 0
    for (int i = 0; i < n; i++) {
	cout << "  test_pack_unpack:";
	for (int j = 0; j < 8; j++)
	    cout << "  " << b.at({i,j});
	cout << "\n";
    }
#endif

    for (int i = 0; i < n; i++)
	assert(a.at({i}) == c.at({i}));

    cout << "test_pack_unpack: pass" << endl;
}


// -------------------------------------------------------------------------------------------------
//
// negate_4bit() testing


__global__ void negate_4bit_kernel(int *buf, int nsites, int ninner)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = gridDim.x * blockDim.x;
    
    for (int i = threadId; i < nsites; i += nthreads) {
	int x = buf[i];
	int y = 0;

	for (int j = 0; j < ninner; j++) {
	    x = CorrelatorKernel<128,32>::negate_4bit(x);
	    y |= x;   // This kludge is needed to prevent the compiler from optimizing away the loop.
	}

	buf[i] = y;
    }
}


__host__ void test_negate_4bit()
{
    const int nsites = 512;
    
    // negate_4bit() assumes its 4-bit input values are in the range [-7,8),
    // i.e. it fails for -8. The loop below generates random input data
    // satisfying this constraint.

    Array<int> src({nsites,8}, af_rhost);
    for (int i = 0; i < nsites; i++)
	for (int j = 0; j < 8; j++)
	    src.at({i,j}) = rand_int(-7, 8);

    Array<int> dst = pack_4bit(src).to_gpu();
    
    negate_4bit_kernel <<<nsites/32, 32>>> (dst.data, nsites, 1);
    CUDA_PEEK("negate_4bit_kernel (test)");
    CUDA_CALL(hipDeviceSynchronize());

    dst = unpack_4bit(dst.to_host());

    for (int i = 0; i < nsites; i++)
	for (int j = 0; j < 8; j++)
	    assert(dst.at({i,j}) == -src.at({i,j}));

    cout << "test_negate_4bit: pass" << endl;
}


// -------------------------------------------------------------------------------------------------
//
// transpose_rank8_4bit() testing


__global__ void transpose_rank8_4bit_kernel(int *buf, int nsites, int ninner)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = gridDim.x * blockDim.x;

    for (int i = threadId; i < nsites; i += nthreads) {
	int x[8];

	#pragma unroll
	for (int j = 0; j < 8; j++)
	    x[j] = buf[8*i + j];

	for (int j = 0; j < ninner; j++)
	    CorrelatorKernel<128,32>::transpose_rank8_4bit(x);

	#pragma unroll
	for (int j = 0; j < 8; j++)
	    buf[8*i+j] = x[j];
    }
}


__host__ void test_transpose_rank8_4bit()
{
    const int nsites = 512;

    Array<int> a({nsites,8}, af_rhost | af_random);
    Array<int> src = unpack_4bit(a);
    
    a = a.to_gpu();
    transpose_rank8_4bit_kernel <<<nsites/32, 32>>> (a.data, nsites, 1);
    CUDA_PEEK("transpose_rank8_4bit_kernel");
    CUDA_CALL(hipDeviceSynchronize());
    
    a = a.to_host();
    Array<int> dst = unpack_4bit(a);

    for (int i = 0; i < nsites; i++)
	for (int j = 0; j < 8; j++)
	    for (int k = 0; k < 8; k++)
		assert(src.at({i,j,k}) == dst.at({i,k,j}));

    cout << "test_transpose_rank8_4bit: pass" << endl;
}


// -------------------------------------------------------------------------------------------------


__host__ int8_t pack_complex44(complex<int> z)
{
    assert((z.real() >= -7) && (z.real() <= 7));
    assert((z.imag() >= -7) && (z.imag() <= 7));

    if constexpr (CorrelatorParams::real_part_in_low_bits)
	return (z.real() & 0xf) | (z.imag() << 4);
    else
	return (z.imag() & 0xf) | (z.real() << 4);
}


__host__ void minimal_correlator_test(int nstations, int nfreq, int f, int sa, int sb, int t, complex<int> za, complex<int> zb, bool rfi_bit=true)
{
    // Currently hardcoded
    const int nt_outer = 4;
    const int nt_inner = 1024;
    
    const int nt_tot = nt_outer * nt_inner;
    const int touter = int(t / nt_inner);
    const int nvtiles = ((nstations/16) * (nstations/16+1)) / 2;
    
    assert(!CorrelatorParams::artificially_remove_input_shuffle);
    assert(!CorrelatorParams::artificially_remove_output_shuffle);
    assert(!CorrelatorParams::artificially_remove_negate_4bit);
    
    assert((f >= 0) && (f < nfreq));
    assert((sa >= 0) && (sa < nstations));
    assert((sb >= 0) && (sb < nstations));
    assert((t >= 0) && (t < nt_outer*nt_inner));
    assert((za.real() >= -7) && (za.real() <= 7));
    assert((za.imag() >= -7) && (za.imag() <= 7));
    assert((zb.real() >= -7) && (zb.real() <= 7));
    assert((zb.imag() >= -7) && (zb.imag() <= 7));
    
    assert(sa >= sb);
    assert((sa > sb) || (za == zb));
    
    int ea = pack_complex44(za);
    int eb = pack_complex44(zb);
    int ahi = sa >> 4;
    int bhi = sb >> 4;
    int alo = sa & 0xf;
    int blo = sb & 0xf;
    int taa = (ahi*(ahi+1))/2 + ahi;
    int tab = (ahi*(ahi+1))/2 + bhi;
    int tbb = (bhi*(bhi+1))/2 + bhi;
    
    cout << "minimal_correlator_test(): start:"
	 << " nstations=" << nstations << ", nfreq=" << nfreq
	 << ", f=" << f << ", sa=" << sa << ", sb=" << sb << ", t=" << t
	 << ", za=" << za << ", Ea=" << ea << ", zb=" << zb << ", Eb=" << eb
	 << ", rfi_bit=" << rfi_bit << endl;

    Correlator corr(nstations, nfreq);
    
    Array<int8_t> emat({nt_tot,nfreq,nstations}, af_rhost | af_zero);
    emat.at({t,f,sa}) = ea;
    emat.at({t,f,sb}) = eb;

    assert((nt_tot % 32) == 0);
    Array<uint> rfimask({nfreq,nt_tot/32}, af_rhost | af_zero);
    Array<int> vmat_cpu({nt_outer,nfreq,nvtiles,16,16,2}, af_rhost | af_zero);
    Array<int> vmat_gpu({nt_outer,nfreq,nvtiles,16,16,2}, af_random | af_gpu);

    // Compute rfimask and vmat_cpu.
    if (rfi_bit) {
	int j = (f*nt_outer*nt_inner + t) / 32;
	rfimask.data[j] = (1U << (t % 32));
	
	vmat_cpu.at({touter,f,taa,alo,alo,0}) = (za * conj(za)).real();
	vmat_cpu.at({touter,f,tbb,blo,blo,0}) = (zb * conj(zb)).real();
	vmat_cpu.at({touter,f,tab,alo,blo,0}) = (za * conj(zb)).real();
	vmat_cpu.at({touter,f,tab,alo,blo,1}) = (za * conj(zb)).imag();
	
	if (ahi == bhi) {
	    vmat_cpu.at({touter,f,tab,blo,alo,0}) = (zb * conj(za)).real();
	    vmat_cpu.at({touter,f,tab,blo,alo,1}) = (zb * conj(za)).imag();
	}
    }

    // Compute vmat_gpu.
    emat = emat.to_gpu();
    rfimask = rfimask.to_gpu();
    corr.launch(vmat_gpu, emat, rfimask, nt_outer, nt_inner, nullptr, true);  // sync=true
    vmat_gpu = vmat_gpu.to_host();

    // Compare results.
    
    int nfail = 0;

    for (int to = 0; to < nt_outer; to++) {
	for (int f = 0; f < nfreq; f++) {
	    for (int vt = 0; vt < nvtiles; vt++) {
		for (int i = 0; i < 16; i++) {
		    for (int k = 0; k < 16; k++) {
			int cpu_re = vmat_cpu.at({to,f,vt,i,k,0});
			int cpu_im = vmat_cpu.at({to,f,vt,i,k,1});
			int gpu_re = vmat_gpu.at({to,f,vt,i,k,0});
			int gpu_im = vmat_gpu.at({to,f,vt,i,k,1});

			if ((cpu_re == 0) && (cpu_im == 0) && (gpu_re == 0) && (gpu_im == 0))
			    continue;

			bool fail = (cpu_re != gpu_re) || (cpu_im != gpu_im);

			cout << "    " << (fail ? "FAILED" : "looks good")
			     << ": at touter=" << to << ", f=" << f << ", vt=" << vt << ", i=" << i << ", k=" << k << ": "
			     << "expected (re,im)=(" << cpu_re << "," << cpu_im << "), "
			     << "got (re,im)=(" << gpu_re << "," << gpu_im << ")" << endl;
			
			if (fail)
			    nfail++;
		
			if (nfail >= 32) {
			    cout << "    Reached threshold failure count; aborting test" << endl;
			    exit(1);
			}
		    }
		}
	    }
	}
    }

    if (nfail > 0) {
	cout << "    Test failed" << endl;
	exit(1);
    }

    cout << "minimal_correlator_test(): pass" << endl;
}


void test_correlator(int nstations, int nfreq, int nt_outer, int nt_inner, int M=10)
{
    const int nt_tot = (nt_outer * nt_inner);
    const int nvtiles = ((nstations/16) * (nstations/16+1)) / 2;
    
    assert(!CorrelatorParams::artificially_remove_input_shuffle);
    assert(!CorrelatorParams::artificially_remove_output_shuffle);
    assert(!CorrelatorParams::artificially_remove_negate_4bit);

    cout << "\ntest_correlator("
	 << "nstations=" << nstations
	 << ", nfreq=" << nfreq
	 << ", nt_outer=" << nt_outer
	 << ", nt_inner=" << nt_inner
	 << ")" << endl;

    Array<int> vmat_cpu({nt_outer,nfreq,nvtiles,16,16,2}, af_rhost | af_zero);
    Array<int8_t> emat({nt_tot,nfreq,nstations}, af_rhost | af_zero);
    
    assert((nt_tot % 32) == 0);
    Array<uint> rfimask({nfreq,nt_tot/32}, af_rhost | af_zero);

    // Randomize rfimask.
    // Note: I checked that gputils::default_rng() returns a uint in which all bits are random.
    
    for (int i = 0; i < rfimask.size; i++)
	rfimask.data[i] = gputils::default_rng();
    
    vector<int> ix(nstations);
    for (int i = 0; i < nstations; i++)
	ix[i] = i;

    vector<complex<int>> z(M);

    for (int touter = 0; touter < nt_outer; touter++) {
	for (int t = touter*nt_inner; t < (touter+1)*nt_inner; t++) {
	    for (int f = 0; f < nfreq; f++) {
		// Generate M random indices in [0:nstations).
		for (int i = 0; i < M; i++) {
		    int j = rand_int(i, nstations);
		    std::swap(ix[i], ix[j]);
		}
	    
		// Generate M random E-array values.
		for (int i = 0; i < M; i++) {
		    z[i] = { int(rand_int(-7,8)), int(rand_int(-7,8)) };
		    emat.at({t,f,ix[i]}) = pack_complex44(z[i]);
		}

		// Skip updating vmat_cpu if RFI mask bit is zero.
		uint rm = rfimask.data[(f*nt_tot + t) / 32];
		uint bit = 1U << (t % 32);
		if ((rm & bit) == 0)
		    continue;

		// Update vmat_cpu.
		for (int i = 0; i < M; i++) {
		    int ahi = ix[i] >> 4;
		    int alo = ix[i] & 0xf;
		    
		    for (int j = 0; j < M; j++) {
			int bhi = ix[j] >> 4;
			int blo = ix[j] & 0xf;

			if (ahi < bhi)
			    continue;

			int vt = (ahi*(ahi+1))/2 + bhi;			
			complex<int> zz = z[i] * conj(z[j]);
			vmat_cpu.at({touter,f,vt,alo,blo,0}) += zz.real();
			vmat_cpu.at({touter,f,vt,alo,blo,1}) += zz.imag();
		    }
		}
	    }
	}
    }

    emat = emat.to_gpu();
    rfimask = rfimask.to_gpu();
    Array<int> vmat_gpu({nt_outer,nfreq,nvtiles,16,16,2}, af_random | af_gpu);

    Correlator corr(nstations, nfreq);
    corr.launch(vmat_gpu, emat, rfimask, nt_outer, nt_inner, nullptr, true);  // sync=true
    vmat_gpu = vmat_gpu.to_host();

    for (int touter = 0; touter < nt_outer; touter++) {
	for (int f = 0; f < nfreq; f++) {
	    for (int vt = 0; vt < nvtiles; vt++) {
		for (int i = 0; i < 16; i++) {
		    for (int k = i; k < 16; k++) {
			complex<int> vcpu = complex<int> (vmat_cpu.at({touter,f,vt,i,k,0}), vmat_cpu.at({touter,f,vt,i,k,1}));
			complex<int> vgpu = complex<int> (vmat_gpu.at({touter,f,vt,i,k,0}), vmat_gpu.at({touter,f,vt,i,k,1}));
		    
			if (vcpu != vgpu) {
			    cout << "test_correlator() failed at touter=" << touter
				 << ", f=" << f << ", vt=" << vt << ", i=" << i << ", k=" << k
				 << ": vcpu = " << vcpu << ", vgpu = " << vgpu
				 << endl;
			    
			    exit(1);
			}
		    }
		}
	    }
	}
    }
    
    cout << "test_correlator(): pass" << endl;
}


// -------------------------------------------------------------------------------------------------


int main(int argc, char **argv)
{
    test_pack_unpack();
    test_negate_4bit();
    test_transpose_rank8_4bit();

    // In the "minimal" test, only two entries in the E-array are nonzero, and
    // the test output is verbose. This can be useful for tracking down bugs,
    // but we don't run it by default.

    // (nstations, nfreq, f, sa, sb, t, za, ab)
    // minimal_correlator_test(128, 128, 3, 37, 23, 183, {1,2}, {3,4});
    // minimal_correlator_test(1024, 8, 5, 335, 159, 137, {1,2}, {3,4});

    // Full end-to-end test starts here.
    
    std::mt19937 rng(137);
    const double maxbytes = 2.0e9;  // 2 GB

    // List of pairs (nstations, nfreq)
    vector<pair<int,int>> kparams = get_all_kernel_params();
    
    for (auto p: kparams) {
	int nstations = p.first;
	int nfreq = p.second;
	
	int nvtiles = ((nstations/16) * (nstations/16+1)) / 2;
	double nbytes_e = nfreq * nstations;           // multiply by (nt_inner * nt_outer)
	double nbytes_v = 2048.0 * nfreq * nvtiles;    // multiply by (nt_outer)
	
	int max_multiplier = int((0.9999*maxbytes - nbytes_v) / (256. * nbytes_e));
	int nt_inner = 256 * gputils::rand_int(1, min(max_multiplier,10)+1, rng);
	
	int max_nt_outer = int(maxbytes / (nt_inner*nbytes_e + nbytes_v));
	int nt_outer = gputils::rand_int(1, max_nt_outer+1, rng);
	test_correlator(nstations, nfreq, nt_outer, nt_inner);
    }

    return 0;
}

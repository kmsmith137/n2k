#include "hip/hip_runtime.h"
#include "../include/n2k.hpp"
#include "../include/n2k/interpolation.hpp"

#include <gputils/string_utils.hpp>
#include <iostream>

using namespace std;
using namespace n2k;


static bool is_perfect_square(int n)
{
    if (n < 1)
	return false;
    int m = int(sqrt(n) + 0.5);
    return n == m*m;
}


// FIXME move to libn2k.so (needs GPU-friendly interface)
template<typename T>
struct Bvec
{
    int n = 0;
    double *bias_coeffs = nullptr;
    
    int min_s1 = 0;
    int max_s1 = 0;

    // Length 98*n+1
    vector<T> bvec;

    // Helper function for constructor.
    // No range check on y!
    inline double _eval_y(int ix, double y)
    {
	using sk_globals::nx;
	using sk_globals::ny;

	assert((ix >= 0) && (ix < nx));
	
	double ret = 0.0;
	double ypow = 1.0;
	
	for (int j = 0; j < ny; j++) {
	    ret += bias_coeffs[ix*ny+j] * ypow;
	    ypow *= y;
	}

	return ret;
    }
	
    
    // Helper function for constructor.
    inline double _interpolate(double x, double y)
    {
	using sk_globals::xmin;
	using sk_globals::xmax;
	using sk_globals::nx;

	// Normalize to [0,nx-1]
	double t = (nx-1) * (x-xmin) / (xmax-xmin);
	assert(t >= 0.9);
	assert(t <= nx-1.9);
	
	int it = int(t);
	it = std::max(it,1);
	it = std::min(it,nx-3);

	double b0 = _eval_y(it-1, y);
	double b1 = _eval_y(it, y);
	double b2 = _eval_y(it+1, y);
	double b3 = _eval_y(it+2, y);

	return cubic_interpolate(t-it, b0, b1, b2, b3);
    }
    
    Bvec(int n_, bool verbose=false)
    {
	using sk_globals::mu_min;
	using sk_globals::mu_max;
	using sk_globals::n_min;
	
	this->n = n_;
	this->bias_coeffs = sk_globals::get_bias_coeffs();
	assert(n >= n_min);

	double eps = 5.0e-11;
	this->min_s1 = n * mu_min;
        this->min_s1 = int((1-eps)*min_s1 + 1);  // round up
        this->max_s1 = n * mu_max;
        this->max_s1 = int((1+eps)*max_s1);      // round down

	if (verbose) {
	    cout << "min_s1 = " << min_s1 << endl;
	    cout << "max_s1 = " << max_s1 << endl;
	}
	
	assert(min_s1 > 0);
	assert(min_s1 <= max_s1);
	assert(max_s1 <= 98*n);
	
	this->bvec = vector<T> (98*n+1, 0);

	// FIXME in hindsight, interpolation could be done more efficiently in two
	// steps. First, take the 2-d shape-(128,4) coefficient array, and "evaluate y"
	// to get a 1-d length-128 array. Then, "resample x" by interpolation, to get
	// a 1-d length-(98*n+1) array.

	for (int s1 = min_s1; s1 <= max_s1; s1++) {
	    double mu = s1/double(n);
	    double x = log(mu);
	    double y = 1.0/n;
	    bvec[s1] = _interpolate(x, y);

	    if (verbose)
		cout << "   bvec[" << s1 << "] = " << bvec[s1] << endl;
	}
    }
};



// FIXME move to gputils!
inline int quantize(double x)
{
    x = std::abs(x);
    int i = int(x + 0.5);
    return std::min(i, 7);
}


int main(int argc, char **argv)
{
    std::mt19937 &rng = gputils::default_rng;
    bool verbose = false;
    
    if (argc != 3) {
	cerr << "Usage: test-sk-bias <rms> <n>" << endl;
	return 2;
    }

    double rms = gputils::from_str<double> (argv[1]);
    int n = gputils::from_str<int> (argv[2]);
    cout << "Running Monte Carlos with rms=" << rms << ", n=" << n << endl;

    std::normal_distribution<double> dist(0, rms);
    Bvec<double> bvec(n, verbose);

    long nmc = 0;
    long ndenom = 0;
    double sum_sk = 0.0;
    double sum_sk2 = 0.0;
    int nbatch = (1<<21)/n + 1;

    // Run forever!
    for (int iouter = 1; 1; iouter++) {
	for (int b = 0; b < nbatch; b++) {
	    int s1 = 0;
	    int s2 = 0;

	    for (int i = 0; i < n; i++) {
		int ex = quantize(dist(rng));
		int ey = quantize(dist(rng));
		int e2 = ex*ex + ey*ey;
		s1 += e2;
		s2 += e2*e2;
	    }

	    ndenom++;
	    
	    if ((s1 < bvec.min_s1) || (s1 > bvec.max_s1))
		continue;

	    double ds1 = double(s1);
	    double sk = n*s2/(ds1*ds1) - 1;
	    sk *= (n+1) / double(n-1);
	    sk -= bvec.bvec[s1];

	    nmc++;
	    sum_sk += sk;
	    sum_sk2 += sk*sk;
	}

	if (!is_perfect_square(iouter) || (nmc < 2))
	    continue;

	double mean = sum_sk / nmc;
	double rms = sqrt((sum_sk2 - nmc*mean*mean) / double(nmc-1) / double(nmc));
	double pvalid = nmc / double(ndenom);

	cout << "nmc=" << nmc << ", pvalid=" << pvalid
	     << ", mean=" << mean << ", rms=" << rms << endl;
    }
}

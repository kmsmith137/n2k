#include "hip/hip_runtime.h"
#include "../include/n2k/s012_kernels.hpp"
#include "../include/n2k/internals.hpp"

#include <gputils/cuda_utils.hpp>

using namespace std;
using namespace gputils;


namespace n2k {
#if 0
}  // editor auto-indent
#endif


// Kernel arguments:
//
//   uint   S012_out[T/Nds][M];   // where M is number of "spectator" indices (3*F*S)
//   uint   S012_in[T][M]; 
//   long   Nds;
//   long   Tout;
//   long   M;
//
// Parallelization:
//
//   - Each thread does one (output time, spectator index).
//
//   - Index mapping is:
//
//      {threadIdx,blockIdx}.x <-> spectator index
//      {threadIdx,blockIdx}.y <-> output (downsampled) time

__global__ void s012_time_downsample_kernel(uint *Sout, const uint *Sin, int Nds, int Tout, int M)
{
    // Per-thread (downsampled time, spectator index)
    uint m = (blockIdx.x * blockDim.x) + threadIdx.x;
    ulong tout = (blockIdx.y * blockDim.y) + threadIdx.y;
    bool valid = (m < M) && (tout < Tout);

    // Per-thread base indices and pointer shift.
    ulong in_base = (M*Nds)*tout + m;
    ulong out_base = M*tout + m;
    Sin += (valid ? in_base : 0);
    
    uint s = 0;
    for (int n = 0; n < Nds; n++)
	s += Sin[n*M];
    
    if (valid)
	Sout[out_base] = s;
}

// launch_s012_time_downsample_kernel(): bare-pointer interface.
//
//   uint   S012_out[T/Nds][M];   // where M is number of "spectator" indices (3*F*S)
//   uint   S012_in[T][M];
//   long   Nds;
//   long   Tout;
//   long   M;

void launch_s012_time_downsample_kernel(uint *Sout, const uint *Sin, long Nds, long Tout, long M, hipStream_t stream)
{
    int threads_per_block = 128;
    bool noisy = false;

    if ((Sout == nullptr) || (Sin == nullptr))
	throw runtime_error("launch_s012_time_downsample_kernel(): data pointer was NULL");
    if (Nds <= 0)
	throw runtime_error("launch_s012_time_downsample_kernel(): expected Nds > 0");
    if (Tout <= 0)
	throw runtime_error("launch_s012_time_downsample_kernel(): expected Tout > 0");
    if (M <= 0)
	throw runtime_error("launch_s012_time_downsample_kernel(): expected M > 0");
    
    // Necessary for efficiency (but not correctness) of kernel.
    // Always satisfied for CHORD, CHIME, HIRAX.
    if (M % 32)
	throw runtime_error("launch_s012_time_downsample_kernel(): expected M to be a multple of 32");
    
    dim3 nblocks, nthreads;
    gputils::assign_kernel_dims(nblocks, nthreads, M, Tout, 1, threads_per_block, noisy);

    s012_time_downsample_kernel <<< nblocks, nthreads, 0, stream >>>
	(Sout, Sin, Nds, Tout, M);

    CUDA_PEEK("launch s012_time_downsample");
}


// launch_s012_time_downsample_kernel(): array interface
//
//   uint   S012_out[T/Nds][F][3][S];   // S must be multiple of 32
//   uint   S012_in[T][F][3][S];
//   long   Nds;

void launch_s012_time_downsample_kernel(Array<uint> &Sout, const Array<uint> &Sin, long Nds, hipStream_t stream)
{
    check_array(Sout, "launch_s012_time_downsample_kernel", "Sout", 4, true);  // ndim=4, contiguous=true
    check_array(Sin, "launch_s012_time_downsample_kernel", "Sin", 4, true);    // ndim=4, contiguous=true

    if (Sin.shape[0] != Sout.shape[0] * Nds)
	throw runtime_error("launch_s012_time_downsample_kernel(): inconsistent number of time samples in input/output arrays");
    if (Sin.shape[1] != Sout.shape[1])
	throw runtime_error("launch_s012_time_downsample_kernel(): inconsistent number of frequency channels in input/output arrays");
    if (Sout.shape[2] != 3)
	throw runtime_error("launch_s012_time_downsample_kernel(): expected Sout.shape[2] == 3");
    if (Sin.shape[2] != 3)
	throw runtime_error("launch_s012_time_downsample_kernel(): expected Sin.shape[2] == 3");
    if (Sin.shape[3] != Sout.shape[3])
	throw runtime_error("launch_s012_time_downsample_kernel(): inconsistent number of stations in input/output arrays");

    long Tout = Sout.shape[0];
    long M = 3 * Sout.shape[1] * Sout.shape[3];
    
    launch_s012_time_downsample_kernel(Sout.data, Sin.data, Nds, Tout, M, stream);
}


}  // namespace n2k
